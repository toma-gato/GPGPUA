#include "hip/hip_runtime.h"
#include "to_bench.cuh"

#include "cuda_tools/cuda_error_checking.cuh"

#include <raft/core/device_span.hpp>

#include <rmm/device_uvector.hpp>
#include <rmm/device_scalar.hpp>


template <typename T>
__global__
void kernel_reduce_baseline(raft::device_span<const T> buffer, raft::device_span<T> total)
{
    for (int i = 0; i < buffer.size(); ++i)
        *total.data() += buffer[i];
}

void baseline_reduce(rmm::device_uvector<int>& buffer,
                     rmm::device_scalar<int>& total)
{
	kernel_reduce_baseline<int><<<1, 1, 0, buffer.stream()>>>(
        raft::device_span<int>(buffer.data(), buffer.size()),
        raft::device_span<int>(total.data(), 1));

    CUDA_CHECK_ERROR(hipStreamSynchronize(buffer.stream()));
}

template <typename T>
__device__ void warp_reduce(raft::device_span<T> sdata, unsigned int tid) {
    sdata[tid] += sdata[tid + 32]; __syncwarp();
    sdata[tid] += sdata[tid + 16]; __syncwarp();
    sdata[tid] += sdata[tid + 8]; __syncwarp();
    sdata[tid] += sdata[tid + 4]; __syncwarp();
    sdata[tid] += sdata[tid + 2]; __syncwarp();
    sdata[tid] += sdata[tid + 1]; __syncwarp();
}

template <typename T>
__global__
void kernel_your_reduce(raft::device_span<const T> buffer, raft::device_span<T> total)
{
    // Help: odd size
    // When treating an odd size think about two things
    // 1. How could a thread sum two values and have the second (that we don't want for the odd case) not have any impact on the sum?
    // 2. Once 1. is achived, could we use a fixed even size while still achieving the same 

    // TODO
    // Your reduce code
    extern __shared__ T sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    if (i < buffer.size()) {
        if (i + blockDim.x < buffer.size())
            sdata[tid] = buffer[i] + buffer[i + blockDim.x];
        else
            sdata[tid] = buffer[i];
    }
    else
        sdata[tid] = 0;
    
    __syncthreads();

    for (int s = blockDim.x / 2; s > 32; s /= 2) {
        if (tid < s)
            sdata[tid] += sdata[tid + s];
        __syncthreads();
    }

    warp_reduce(sdata, tid);

    if (tid == 0) total[blockIdx.x] = sdata[0];
}

void your_reduce(rmm::device_uvector<int>& buffer,
                 rmm::device_scalar<int>& total)
{
    // Help: more than 1 thread block
    // When treating the 2 thread block case you need to create a temporary array
    // To do so use the following API : rmm::device_uvector<int> tmp(<SIZE>, buffer.stream())

    // Help: very large case
    // Using only 2 kernels, what is the biggest buffer size we can handle?

    // TODO fill in blocks, threads, and shared memory
    // Help: To properly compute the amount of block, use the following API: (<PROBLEM_SIZE> + <BLOCK_SIZE> - 1) / <BLOCK_SIZE>

    rmm::device_uvector<int> tmp(1024, buffer.stream());

    kernel_your_reduce<int><<<1024, 1024, 1024 * sizeof(int), buffer.stream()>>>(
        raft::device_span<const int>(buffer.data(), buffer.size()),
        raft::device_span<int>(tmp.data(), 1));

    kernel_your_reduce<int><<<1, 1024, 1024 * sizeof(int), buffer.stream()>>>(
        raft::device_span<const int>(tmp.data(), tmp.size()),
        raft::device_span<int>(total.data(), 1));

    CUDA_CHECK_ERROR(hipStreamSynchronize(buffer.stream()));
}