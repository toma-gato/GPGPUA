#include "hip/hip_runtime.h"
#include "to_bench.cuh"

#include "cuda_tools/cuda_error_checking.cuh"

#include <raft/core/device_span.hpp>

#include <rmm/device_uvector.hpp>

template <typename T>
__global__
void kernel_scan_baseline(raft::device_span<T> buffer)
{
    for (int i = 1; i < buffer.size(); ++i)
        buffer[i] += buffer[i - 1];
}

void baseline_scan(rmm::device_uvector<int>& buffer)
{
	kernel_scan_baseline<int><<<1, 1, 0, buffer.stream()>>>(
        raft::device_span<int>(buffer.data(), buffer.size()));

    CUDA_CHECK_ERROR(hipStreamSynchronize(buffer.stream()));
}

inline __device__ int warp_reduce(int val) {
    #pragma unroll
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(~0, val, offset);
    }
    return val;
}

template <typename T, int BLOCK_SIZE>
__global__
void kernel_your_reduce(raft::device_span<const T> buffer, raft::device_span<T> total)
{
    extern __shared__ T sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    unsigned int gridSize = BLOCK_SIZE * 2 * gridDim.x;

    sdata[tid] = 0;
    while(i < buffer.size()) {
        if (i + BLOCK_SIZE < buffer.size())
            sdata[tid] += buffer[i] + buffer[i + BLOCK_SIZE];
        else
            sdata[tid] += buffer[i];
        i += gridSize;
    }
    
    __syncthreads();

    if constexpr (BLOCK_SIZE >= 1024) {
        if (tid < 512)
            sdata[tid] += sdata[tid + 512];
        __syncthreads();
    }
    if constexpr (BLOCK_SIZE >= 512) {
        if (tid < 256)
            sdata[tid] += sdata[tid + 256];
        __syncthreads();
    }
    if constexpr (BLOCK_SIZE >= 256) {
        if (tid < 128)
            sdata[tid] += sdata[tid + 128];
        __syncthreads();
    }
    if constexpr (BLOCK_SIZE >= 128) {
        if (tid < 64)
            sdata[tid] += sdata[tid + 64];
        __syncthreads();
    }
    if constexpr (BLOCK_SIZE >= 64) {
        if (tid < 32)
            sdata[tid] += sdata[tid + 32];
        __syncthreads();
    }

    if (tid < 32)
        sdata[tid] += warp_reduce(sdata[tid]);

    if (tid == 0) total[blockIdx.x] = sum;
}


template <typename T>
__global__
void kernel_your_scan(raft::device_span<T> buffer)
{
    // TODO
    unsigned int tid = threadIdx.x;
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = 1; i < buffer.size(); i*=2) {
        T val = 0;
        if (tid >= i) {
            val = buffer[idx - i];
        }
        __syncthreads();

        buffer[idx] += val;
        __syncthreads();
    }
}

void your_scan(rmm::device_uvector<int>& buffer)
{
    // TODO
    // rmm::device_uvector<int> tmp(2, buffer.stream());

    // kernel_your_reduce<int, 64><<<2, 64, 0, buffer.stream()>>>(
    //     raft::device_span<const int>(buffer.data(), buffer.size()),
    //     raft::device_span<int>(tmp.data(), 1));

	kernel_your_scan<int><<<1, 64, 0, buffer.stream()>>>(
        raft::device_span<int>(buffer.data(), buffer.size()));

    

    CUDA_CHECK_ERROR(hipStreamSynchronize(buffer.stream()));
}