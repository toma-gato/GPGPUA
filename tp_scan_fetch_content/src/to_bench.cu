#include "hip/hip_runtime.h"
#include "to_bench.cuh"

#include "cuda_tools/cuda_error_checking.cuh"

#include <raft/core/device_span.hpp>

#include <rmm/device_uvector.hpp>

template <typename T>
__global__
void kernel_scan_baseline(raft::device_span<T> buffer)
{
    for (int i = 1; i < buffer.size(); ++i)
        buffer[i] += buffer[i - 1];
}

void baseline_scan(rmm::device_uvector<int>& buffer)
{
	kernel_scan_baseline<int><<<1, 1, 0, buffer.stream()>>>(
        raft::device_span<int>(buffer.data(), buffer.size()));

    CUDA_CHECK_ERROR(hipStreamSynchronize(buffer.stream()));
}

template <typename T>
__global__
void kernel_your_scan(raft::device_span<T> buffer)
{
    // TODO
    // ...
}

void your_scan(rmm::device_uvector<int>& buffer)
{
    // TODO
    // ...

	kernel_your_scan<int><<<1, 1, 0, buffer.stream()>>>(
        raft::device_span<int>(buffer.data(), buffer.size()));

    CUDA_CHECK_ERROR(hipStreamSynchronize(buffer.stream()));
}